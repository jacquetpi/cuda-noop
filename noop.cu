#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <iostream>

__global__ void noop_kernel(int iters) {
    __global__ void noop_kernel(int iters) {
        // A simple busy-wait loop to simulate delay
        for (int i = 0; i < iters; ++i) {
            // Simulated delay (busy-wait)
            volatile int x = 0;
            for (int j = 0; j < 100; ++j) {
                x++;
            }
        }
    }
}

int main() {
    dim3 grid(1);
    dim3 block(32); // 1 warp
    int iters = 10000; // 10ms target for kernels

    while (true) {
        noop_kernel<<<grid, block>>>(iters);
        hipDeviceSynchronize();

        usleep(1000); // 1ms pause
    }

    return 0;
}
