#include <hip/hip_runtime.h>
#include <unistd.h>
#include <iostream>

__global__ void noop_kernel(int iters) {
    for (int i = 0; i < iters; ++i) {
        __nanosleep(100); // Supported on Volta and higher
    }
}

int main() {
    dim3 grid(1);
    dim3 block(32); // 1 warp
    int iters = 10000; // 10ms target for kernels

    while (true) {
        noop_kernel<<<grid, block>>>(iters);
        hipDeviceSynchronize();

        usleep(1000); // 1ms pause
    }

    return 0;
}
